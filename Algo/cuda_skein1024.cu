#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <>
#include <inttypes.h>
#include <stdio.h>
#include <memory.h>

typedef unsigned char uint8_t;
typedef unsigned short uint16_t;
typedef unsigned int uint32_t;
#if __WORDSIZE == 64
typedef unsigned long uint64_t;
#else
typedef unsigned long long uint64_t;
#endif

#include "cuda_helper.h"

extern int device_major[8];
extern int device_minor[8];

extern int device_map[8];

extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

__constant__ uint64_t pTarget[16];

#define ROL64(x, n)        (((x) << (n)) | ((x) >> (64 - (n))))

static __constant__ uint2 uMessage[27];
static __constant__ uint2 c_hv[17];
static __constant__ uint2 skein_ks_parity = { 0x55555555, 0x55555555 };

static __constant__ uint2 t12[9] =
{
	{ 0x80, 0 },
	{ 0, 0x70000000 },
	{ 0x80, 0x70000000 },
	{ 0xd8, 0 },
	{ 0, 0xb0000000 },
	{ 0xd8, 0xb0000000 },
	{ 0x08, 0 },
	{ 0, 0xff000000 },
	{ 0x08, 0xff000000 }
};

static const uint64_t cpu_SKEIN1024_IV_1024[16] =
{
	0x5A4352BE62092156,
	0x5F6E8B1A72F001CA,
	0xFFCBFE9CA1A2CE26,
	0x6C23C39667038BCA,
	0x583A8BFCCE34EB6C,
	0x3FDBFB11D4A46A3E,
	0x3304ACFCA8300998,
	0xB2F6675FA17F0FD2,
	0x9D2599730EF7AB6B,
	0x0914A20D3DFEA9E4,
	0xCC1A9CAFA494DBD3,
	0x9828030DA0A6388C,
	0x0D339D5DAADEE3DC,
	0xFC46DE35C4E2A086,
	0x53D6E4F52E19A6D1,
	0x5663952F715D1DDD,
};
static const int cpu_ROT1024[8][8] =
{
	{ 55, 43, 37, 40, 16, 22, 38, 12 },
	{ 25, 25, 46, 13, 14, 13, 52, 57 },
	{ 33, 8, 18, 57, 21, 12, 32, 54 },
	{ 34, 43, 25, 60, 44, 9, 59, 34 },
	{ 28, 7, 47, 48, 51, 9, 35, 41 },
	{ 17, 6, 18, 25, 43, 42, 40, 15 },
	{ 58, 7, 32, 45, 19, 18, 2, 56 },
	{ 47, 49, 27, 58, 37, 48, 53, 56 }
};

static __forceinline__ __device__ void Round1024(uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7,
	uint2 &p8, uint2 &p9, uint2 &pA, uint2 &pB, uint2 &pC, uint2 &pD, uint2 &pE, uint2 &pF,
	int r0, int r1, int r2, int r3, int r4, int r5, int r6, int r7) {
	p0 += p1;
	p2 += p3;
	p4 += p5;
	p6 += p7;
	p8 += p9;
	pA += pB;
	pC += pD;
	pE += pF;

	p1 = ROL2(p1, r0) ^ p0;
	p3 = ROL2(p3, r1) ^ p2;
	p5 = ROL2(p5, r2) ^ p4;
	p7 = ROL2(p7, r3) ^ p6;
	p9 = ROL2(p9, r4) ^ p8;
	pB = ROL2(pB, r5) ^ pA;
	pD = ROL2(pD, r6) ^ pC;
	pF = ROL2(pF, r7) ^ pE;
}

static __forceinline__ __host__ void Round1024_host(uint64_t &p0, uint64_t &p1, uint64_t &p2, uint64_t &p3, uint64_t &p4, uint64_t &p5, uint64_t &p6, uint64_t &p7,
	uint64_t &p8, uint64_t &p9, uint64_t &pA, uint64_t &pB, uint64_t &pC, uint64_t &pD, uint64_t &pE, uint64_t &pF, int ROT)
{
	p0 += p1;
	p1 = ROL64(p1, cpu_ROT1024[ROT][0]);
	p1 ^= p0;
	p2 += p3;
	p3 = ROL64(p3, cpu_ROT1024[ROT][1]);
	p3 ^= p2;
	p4 += p5;
	p5 = ROL64(p5, cpu_ROT1024[ROT][2]);
	p5 ^= p4;
	p6 += p7;
	p7 = ROL64(p7, cpu_ROT1024[ROT][3]);
	p7 ^= p6;
	p8 += p9;
	p9 = ROL64(p9, cpu_ROT1024[ROT][4]);
	p9 ^= p8;
	pA += pB;
	pB = ROL64(pB, cpu_ROT1024[ROT][5]);
	pB ^= pA;
	pC += pD;
	pD = ROL64(pD, cpu_ROT1024[ROT][6]);
	pD ^= pC;
	pE += pF;
	pF = ROL64(pF, cpu_ROT1024[ROT][7]);
	pF ^= pE;
}


uint64_t *d_sknounce[8];
uint64_t *d_SKNonce[8];

__device__ __forceinline__
uint2 ROL8(const uint2 a){
	uint2 result;
	result.x = __byte_perm(a.x, a.y, 0x2107);
	result.y = __byte_perm(a.y, a.x, 0x2107);
	return result;
}

__device__ __forceinline__
uint2 ROR8(const uint2 a){
	uint2 result;
	result.x = __byte_perm(a.x, a.y, 0x4321);
	result.y = __byte_perm(a.y, a.x, 0x4321);
	return result;
}

__constant__ uint2 keccak_round_constants[24] = {
	{ 0x00000001ul, 0x00000000 }, { 0x00008082ul, 0x00000000 },
	{ 0x0000808aul, 0x80000000 }, { 0x80008000ul, 0x80000000 },
	{ 0x0000808bul, 0x00000000 }, { 0x80000001ul, 0x00000000 },
	{ 0x80008081ul, 0x80000000 }, { 0x00008009ul, 0x80000000 },
	{ 0x0000008aul, 0x00000000 }, { 0x00000088ul, 0x00000000 },
	{ 0x80008009ul, 0x00000000 }, { 0x8000000aul, 0x00000000 },
	{ 0x8000808bul, 0x00000000 }, { 0x0000008bul, 0x80000000 },
	{ 0x00008089ul, 0x80000000 }, { 0x00008003ul, 0x80000000 },
	{ 0x00008002ul, 0x80000000 }, { 0x00000080ul, 0x80000000 },
	{ 0x0000800aul, 0x00000000 }, { 0x8000000aul, 0x80000000 },
	{ 0x80008081ul, 0x80000000 }, { 0x00008080ul, 0x80000000 },
	{ 0x80000001ul, 0x00000000 }, { 0x80008008ul, 0x80000000 }
};

#define bitselect(a, b, c) ((a) ^ ((c) & ((b) ^ (a))))

static void __forceinline__ __device__ keccak_1600(uint2 *s)
{
	uint2 bc[5], tmpxor[5], tmp1, tmp2;

	#pragma unroll 2
	for (int i = 0; i < 24; i++)
	{
		#pragma unroll
		for (uint32_t x = 0; x < 5; x++)
			tmpxor[x] = s[x] ^ s[x + 5] ^ s[x + 10] ^ s[x + 15] ^ s[x + 20];

		bc[0] = tmpxor[0] ^ ROL2(tmpxor[2], 1);
		bc[1] = tmpxor[1] ^ ROL2(tmpxor[3], 1);
		bc[2] = tmpxor[2] ^ ROL2(tmpxor[4], 1);
		bc[3] = tmpxor[3] ^ ROL2(tmpxor[0], 1);
		bc[4] = tmpxor[4] ^ ROL2(tmpxor[1], 1);

		tmp1 = s[1] ^ bc[0];

		s[0] ^= bc[4];
		s[1] = ROL2(s[6] ^ bc[0], 44);
		s[6] = ROL2(s[9] ^ bc[3], 20);
		s[9] = ROL2(s[22] ^ bc[1], 61);
		s[22] = ROL2(s[14] ^ bc[3], 39);
		s[14] = ROL2(s[20] ^ bc[4], 18);
		s[20] = ROL2(s[2] ^ bc[1], 62);
		s[2] = ROL2(s[12] ^ bc[1], 43);
		s[12] = ROL2(s[13] ^ bc[2], 25);
		s[13] = ROL8(s[19] ^ bc[3]);
		s[19] = ROR8(s[23] ^ bc[2]);
		s[23] = ROL2(s[15] ^ bc[4], 41);
		s[15] = ROL2(s[4] ^ bc[3], 27);
		s[4] = ROL2(s[24] ^ bc[3], 14);
		s[24] = ROL2(s[21] ^ bc[0], 2);
		s[21] = ROL2(s[8] ^ bc[2], 55);
		s[8] = ROL2(s[16] ^ bc[0], 45);
		s[16] = ROL2(s[5] ^ bc[4], 36);
		s[5] = ROL2(s[3] ^ bc[2], 28);
		s[3] = ROL2(s[18] ^ bc[2], 21);
		s[18] = ROL2(s[17] ^ bc[1], 15);
		s[17] = ROL2(s[11] ^ bc[0], 10);
		s[11] = ROL2(s[7] ^ bc[1], 6);
		s[7] = ROL2(s[10] ^ bc[4], 3);
		s[10] = ROL2(tmp1, 1);

		tmp1 = s[0]; tmp2 = s[1]; s[0] = bitselect(s[0] ^ s[2], s[0], s[1]); s[1] = bitselect(s[1] ^ s[3], s[1], s[2]); s[2] = bitselect(s[2] ^ s[4], s[2], s[3]); s[3] = bitselect(s[3] ^ tmp1, s[3], s[4]); s[4] = bitselect(s[4] ^ tmp2, s[4], tmp1);
		tmp1 = s[5]; tmp2 = s[6]; s[5] = bitselect(s[5] ^ s[7], s[5], s[6]); s[6] = bitselect(s[6] ^ s[8], s[6], s[7]); s[7] = bitselect(s[7] ^ s[9], s[7], s[8]); s[8] = bitselect(s[8] ^ tmp1, s[8], s[9]); s[9] = bitselect(s[9] ^ tmp2, s[9], tmp1);
		tmp1 = s[10]; tmp2 = s[11]; s[10] = bitselect(s[10] ^ s[12], s[10], s[11]); s[11] = bitselect(s[11] ^ s[13], s[11], s[12]); s[12] = bitselect(s[12] ^ s[14], s[12], s[13]); s[13] = bitselect(s[13] ^ tmp1, s[13], s[14]); s[14] = bitselect(s[14] ^ tmp2, s[14], tmp1);
		tmp1 = s[15]; tmp2 = s[16]; s[15] = bitselect(s[15] ^ s[17], s[15], s[16]); s[16] = bitselect(s[16] ^ s[18], s[16], s[17]); s[17] = bitselect(s[17] ^ s[19], s[17], s[18]); s[18] = bitselect(s[18] ^ tmp1, s[18], s[19]); s[19] = bitselect(s[19] ^ tmp2, s[19], tmp1);
		tmp1 = s[20]; tmp2 = s[21]; s[20] = bitselect(s[20] ^ s[22], s[20], s[21]); s[21] = bitselect(s[21] ^ s[23], s[21], s[22]); s[22] = bitselect(s[22] ^ s[24], s[22], s[23]); s[23] = bitselect(s[23] ^ tmp1, s[23], s[24]); s[24] = bitselect(s[24] ^ tmp2, s[24], tmp1);
		s[0] ^= keccak_round_constants[i];
	}
}

static __forceinline__ __device__ void Round1024_0(uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7,
	uint2 &p8, uint2 &p9, uint2 &pA, uint2 &pB, uint2 &pC, uint2 &pD, uint2 &pE, uint2 &pF, int ROT)
{
	p0 += p1;
	p1 = ROL2(p1, 55) ^ p0;
	p2 += p3;
	p3 = ROL2(p3, 43) ^ p2;
	p4 += p5;
	p5 = ROL2(p5, 37) ^ p4;
	p6 += p7;
	p7 = ROL2(p7, 40) ^ p6;
	p8 += p9;
	p9 = ROL2(p9, 16) ^ p8;
	pA += pB;
	pB = ROL2(pB, 22) ^ pA;
	pC += pD;
	pD = ROL2(pD, 38) ^ pC;
	pE += pF;
	pF = ROL2(pF, 12) ^ pE;
}

static __forceinline__ __device__ void Round1024_1(uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7,
	uint2 &p8, uint2 &p9, uint2 &pA, uint2 &pB, uint2 &pC, uint2 &pD, uint2 &pE, uint2 &pF, int ROT)
{
	p0 += p1;
	p1 = ROL2(p1, 25) ^ p0;
	p2 += p3;
	p3 = ROL2(p3, 25) ^ p2;
	p4 += p5;
	p5 = ROL2(p5, 46) ^ p4;
	p6 += p7;
	p7 = ROL2(p7, 13) ^ p6;
	p8 += p9;
	p9 = ROL2(p9, 14) ^ p8;
	pA += pB;
	pB = ROL2(pB, 13) ^ pA;
	pC += pD;
	pD = ROL2(pD, 52) ^ pC;
	pE += pF;
	pF = ROL2(pF, 57) ^ pE;
}

static __forceinline__ __device__ void Round1024_2(uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7,
	uint2 &p8, uint2 &p9, uint2 &pA, uint2 &pB, uint2 &pC, uint2 &pD, uint2 &pE, uint2 &pF, int ROT)
{
	p0 += p1;
	p1 = ROL2(p1, 33) ^ p0;
	p2 += p3;
	p3 = ROL2(p3, 8) ^ p2;
	p4 += p5;
	p5 = ROL2(p5, 18) ^ p4;
	p6 += p7;
	p7 = ROL2(p7, 57) ^ p6;
	p8 += p9;
	p9 = ROL2(p9, 21) ^ p8;
	pA += pB;
	pB = ROL2(pB, 12) ^ pA;
	pC += pD;
	pD = ROL2(pD, 32) ^ pC;
	pE += pF;
	pF = ROL2(pF, 54) ^ pE;
}

static __forceinline__ __device__ void Round1024_3(uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7,
	uint2 &p8, uint2 &p9, uint2 &pA, uint2 &pB, uint2 &pC, uint2 &pD, uint2 &pE, uint2 &pF, int ROT)
{
	p0 += p1;
	p1 = ROL2(p1, 34) ^ p0;
	p2 += p3;
	p3 = ROL2(p3, 43) ^ p2;
	p4 += p5;
	p5 = ROL2(p5, 25) ^ p4;
	p6 += p7;
	p7 = ROL2(p7, 60) ^ p6;
	p8 += p9;
	p9 = ROL2(p9, 44) ^ p8;
	pA += pB;
	pB = ROL2(pB, 9) ^ pA;
	pC += pD;
	pD = ROL2(pD, 59) ^ pC;
	pE += pF;
	pF = ROL2(pF, 34) ^ pE;
}

static __forceinline__ __device__ void Round1024_4(uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7,
	uint2 &p8, uint2 &p9, uint2 &pA, uint2 &pB, uint2 &pC, uint2 &pD, uint2 &pE, uint2 &pF, int ROT)
{
	p0 += p1;
	p1 = ROL2(p1, 28) ^ p0;
	p2 += p3;
	p3 = ROL2(p3, 7) ^ p2;
	p4 += p5;
	p5 = ROL2(p5, 47) ^ p4;
	p6 += p7;
	p7 = ROL2(p7, 48) ^ p6;
	p8 += p9;
	p9 = ROL2(p9, 51) ^ p8;
	pA += pB;
	pB = ROL2(pB, 9) ^ pA;
	pC += pD;
	pD = ROL2(pD, 35) ^ pC;
	pE += pF;
	pF = ROL2(pF, 41) ^ pE;
}

static __forceinline__ __device__ void Round1024_5(uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7,
	uint2 &p8, uint2 &p9, uint2 &pA, uint2 &pB, uint2 &pC, uint2 &pD, uint2 &pE, uint2 &pF, int ROT)
{
	p0 += p1;
	p1 = ROL2(p1, 17) ^ p0;
	p2 += p3;
	p3 = ROL2(p3, 6) ^ p2;
	p4 += p5;
	p5 = ROL2(p5, 18) ^ p4;
	p6 += p7;
	p7 = ROL2(p7, 25) ^ p6;
	p8 += p9;
	p9 = ROL2(p9, 43) ^ p8;
	pA += pB;
	pB = ROL2(pB, 42) ^ pA;
	pC += pD;
	pD = ROL2(pD, 40) ^ pC;
	pE += pF;
	pF = ROL2(pF, 15) ^ pE;
}

static __forceinline__ __device__ void Round1024_6(uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7,
	uint2 &p8, uint2 &p9, uint2 &pA, uint2 &pB, uint2 &pC, uint2 &pD, uint2 &pE, uint2 &pF, int ROT)
{
	p0 += p1;
	p1 = ROL2(p1, 58) ^ p0;
	p2 += p3;
	p3 = ROL2(p3, 7) ^ p2;
	p4 += p5;
	p5 = ROL2(p5, 32) ^ p4;
	p6 += p7;
	p7 = ROL2(p7, 45) ^ p6;
	p8 += p9;
	p9 = ROL2(p9, 19) ^ p8;
	pA += pB;
	pB = ROL2(pB, 18) ^ pA;
	pC += pD;
	pD = ROL2(pD, 2) ^ pC;
	pE += pF;
	pF = ROL2(pF, 56) ^ pE;
}

static __forceinline__ __device__ void Round1024_7(uint2 &p0, uint2 &p1, uint2 &p2, uint2 &p3, uint2 &p4, uint2 &p5, uint2 &p6, uint2 &p7,
	uint2 &p8, uint2 &p9, uint2 &pA, uint2 &pB, uint2 &pC, uint2 &pD, uint2 &pE, uint2 &pF, int ROT)
{
	p0 += p1;
	p1 = ROL2(p1, 47) ^ p0;
	p2 += p3;
	p3 = ROL2(p3, 49) ^ p2;
	p4 += p5;
	p5 = ROL2(p5, 27) ^ p4;
	p6 += p7;
	p7 = ROL2(p7, 58) ^ p6;
	p8 += p9;
	p9 = ROL2(p9, 37) ^ p8;
	pA += pB;
	pB = ROL2(pB, 48) ^ pA;
	pC += pD;
	pD = ROL2(pD, 53) ^ pC;
	pE += pF;
	pF = ROL2(pF, 56) ^ pE;
}

//__launch_bounds__(512)
//__launch_bounds__(576)
//__launch_bounds__(640)
//__launch_bounds__(704)
//__launch_bounds__(768)
//__launch_bounds__(832)
//__launch_bounds__(896)
//__launch_bounds__(1024)
__launch_bounds__(896) /* 896 performs best */  //Should match parameter in MinerThread.cpp
__global__ void  skein1024_gpu_hash_35(int threads, uint64_t startNonce, uint64_t *resNounce)
{
	//GTX 1060 SC
	//240 MH/s for Skein (2 rounds)
	//153 MH/s for Keccak (3 rounds), 181 MH/s for Prod (3 rounds).

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads)
	{
		uint64_t nonce = startNonce + (uint64_t)thread;

		__align__(16) uint2 h[17];
		__align__(16) uint2 t[3];
		__align__(16) uint2 p[16];
		__align__(16) uint2 state[25];
		__align__(16) uint2 tempnonce = vectorize(nonce);

		

		p[0] = uMessage[16] + c_hv[0];
		p[1] = uMessage[17] + c_hv[1];
		p[2] = uMessage[18] + c_hv[2];
		p[3] = uMessage[19] + c_hv[3];
		p[4] = uMessage[20] + c_hv[4];
		p[5] = uMessage[21] + c_hv[5];
		p[6] = uMessage[22] + c_hv[6];
		p[7] = uMessage[23] + c_hv[7];
		p[8] = uMessage[24] + c_hv[8];
		p[9] = uMessage[25] + c_hv[9];

		

		p[10] = tempnonce + c_hv[10];

		t[0] = t12[3]; // ptr  
		t[1] = t12[4]; // etype
		t[2] = t12[5];

		p[11] = c_hv[11];
		p[12] = c_hv[12];
		p[13] = c_hv[13] + t[0];
		p[14] = c_hv[14] + t[1];
		p[15] = c_hv[15];

		//========================================================================================

		#pragma unroll
		for (int i = 1; i < 21; i += 2)
		{
			Round1024_0(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], p[8], p[9], p[10], p[11], p[12], p[13], p[14], p[15], 0);
			Round1024_1(p[0], p[9], p[2], p[13], p[6], p[11], p[4], p[15], p[10], p[7], p[12], p[3], p[14], p[5], p[8], p[1], 1);
			Round1024_2(p[0], p[7], p[2], p[5], p[4], p[3], p[6], p[1], p[12], p[15], p[14], p[13], p[8], p[11], p[10], p[9], 2);
			Round1024_3(p[0], p[15], p[2], p[11], p[6], p[13], p[4], p[9], p[14], p[1], p[8], p[5], p[10], p[3], p[12], p[7], 3);

			p[0] += c_hv[(i + 0) % 17];
			p[1] += c_hv[(i + 1) % 17];
			p[2] += c_hv[(i + 2) % 17];
			p[3] += c_hv[(i + 3) % 17];
			p[4] += c_hv[(i + 4) % 17];
			p[5] += c_hv[(i + 5) % 17];
			p[6] += c_hv[(i + 6) % 17];
			p[7] += c_hv[(i + 7) % 17];
			p[8] += c_hv[(i + 8) % 17];
			p[9] += c_hv[(i + 9) % 17];
			p[10] += c_hv[(i + 10) % 17];
			p[11] += c_hv[(i + 11) % 17];
			p[12] += c_hv[(i + 12) % 17];
			p[13] += c_hv[(i + 13) % 17] + t[(i + 0) % 3];
			p[14] += c_hv[(i + 14) % 17] + t[(i + 1) % 3];
			p[15] += c_hv[(i + 15) % 17] + make_uint2(i, 0);


			Round1024_4(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], p[8], p[9], p[10], p[11], p[12], p[13], p[14], p[15], 4);
			Round1024_5(p[0], p[9], p[2], p[13], p[6], p[11], p[4], p[15], p[10], p[7], p[12], p[3], p[14], p[5], p[8], p[1], 5);
			Round1024_6(p[0], p[7], p[2], p[5], p[4], p[3], p[6], p[1], p[12], p[15], p[14], p[13], p[8], p[11], p[10], p[9], 6);
			Round1024_7(p[0], p[15], p[2], p[11], p[6], p[13], p[4], p[9], p[14], p[1], p[8], p[5], p[10], p[3], p[12], p[7], 7);

			p[0] += c_hv[(i + 1) % 17];
			p[1] += c_hv[(i + 2) % 17];
			p[2] += c_hv[(i + 3) % 17];
			p[3] += c_hv[(i + 4) % 17];
			p[4] += c_hv[(i + 5) % 17];
			p[5] += c_hv[(i + 6) % 17];
			p[6] += c_hv[(i + 7) % 17];
			p[7] += c_hv[(i + 8) % 17];
			p[8] += c_hv[(i + 9) % 17];
			p[9] += c_hv[(i + 10) % 17];
			p[10] += c_hv[(i + 11) % 17];
			p[11] += c_hv[(i + 12) % 17];
			p[12] += c_hv[(i + 13) % 17];
			p[13] += c_hv[(i + 14) % 17] + t[(i + 1) % 3];
			p[14] += c_hv[(i + 15) % 17] + t[(i + 2) % 3];
			p[15] += c_hv[(i + 16) % 17] + make_uint2(i + 1, 0);
		}

		p[0] ^= uMessage[16];
		p[1] ^= uMessage[17];
		p[2] ^= uMessage[18];
		p[3] ^= uMessage[19];
		p[4] ^= uMessage[20];
		p[5] ^= uMessage[21];
		p[6] ^= uMessage[22];
		p[7] ^= uMessage[23];
		p[8] ^= uMessage[24];
		p[9] ^= uMessage[25];
		p[10] ^= tempnonce;

		h[0] = p[0];
		h[1] = p[1];
		h[2] = p[2];
		h[3] = p[3];
		h[4] = p[4];
		h[5] = p[5];
		h[6] = p[6];
		h[7] = p[7];
		h[8] = p[8];
		h[9] = p[9];
		h[10] = p[10];
		h[11] = p[11];
		h[12] = p[12];
		h[13] = p[13];
		h[14] = p[14];
		h[15] = p[15];
		h[16] = skein_ks_parity;

		#pragma unroll
		for (int i = 0; i<16; i++) h[16] ^= p[i];

		t[0] = t12[6];
		t[1] = t12[7];
		t[2] = t12[8];

		p[13] += t[0];
		p[14] += t[1];

		//========================================================================================

		#pragma unroll
		for (int i = 1; i < 21; i += 2)
		{
			Round1024_0(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], p[8], p[9], p[10], p[11], p[12], p[13], p[14], p[15], 0);
			Round1024_1(p[0], p[9], p[2], p[13], p[6], p[11], p[4], p[15], p[10], p[7], p[12], p[3], p[14], p[5], p[8], p[1], 1);
			Round1024_2(p[0], p[7], p[2], p[5], p[4], p[3], p[6], p[1], p[12], p[15], p[14], p[13], p[8], p[11], p[10], p[9], 2);
			Round1024_3(p[0], p[15], p[2], p[11], p[6], p[13], p[4], p[9], p[14], p[1], p[8], p[5], p[10], p[3], p[12], p[7], 3);

			p[0] += h[(i + 0) % 17];
			p[1] += h[(i + 1) % 17];
			p[2] += h[(i + 2) % 17];
			p[3] += h[(i + 3) % 17];
			p[4] += h[(i + 4) % 17];
			p[5] += h[(i + 5) % 17];
			p[6] += h[(i + 6) % 17];
			p[7] += h[(i + 7) % 17];
			p[8] += h[(i + 8) % 17];
			p[9] += h[(i + 9) % 17];
			p[10] += h[(i + 10) % 17];
			p[11] += h[(i + 11) % 17];
			p[12] += h[(i + 12) % 17];
			p[13] += h[(i + 13) % 17] + t[(i + 0) % 3];
			p[14] += h[(i + 14) % 17] + t[(i + 1) % 3];
			p[15] += h[(i + 15) % 17] + make_uint2(i, 0);


			Round1024_4(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], p[8], p[9], p[10], p[11], p[12], p[13], p[14], p[15], 4);
			Round1024_5(p[0], p[9], p[2], p[13], p[6], p[11], p[4], p[15], p[10], p[7], p[12], p[3], p[14], p[5], p[8], p[1], 5);
			Round1024_6(p[0], p[7], p[2], p[5], p[4], p[3], p[6], p[1], p[12], p[15], p[14], p[13], p[8], p[11], p[10], p[9], 6);
			Round1024_7(p[0], p[15], p[2], p[11], p[6], p[13], p[4], p[9], p[14], p[1], p[8], p[5], p[10], p[3], p[12], p[7], 7);

			p[0] += h[(i + 1) % 17];
			p[1] += h[(i + 2) % 17];
			p[2] += h[(i + 3) % 17];
			p[3] += h[(i + 4) % 17];
			p[4] += h[(i + 5) % 17];
			p[5] += h[(i + 6) % 17];
			p[6] += h[(i + 7) % 17];
			p[7] += h[(i + 8) % 17];
			p[8] += h[(i + 9) % 17];
			p[9] += h[(i + 10) % 17];
			p[10] += h[(i + 11) % 17];
			p[11] += h[(i + 12) % 17];
			p[12] += h[(i + 13) % 17];
			p[13] += h[(i + 14) % 17] + t[(i + 1) % 3];
			p[14] += h[(i + 15) % 17] + t[(i + 2) % 3];
			p[15] += h[(i + 16) % 17] + make_uint2(i + 1, 0);
		}

		//========================================================================================

		state[0] = p[0];
		state[1] = p[1];
		state[2] = p[2];
		state[3] = p[3];
		state[4] = p[4];
		state[5] = p[5];
		state[6] = p[6];
		state[7] = p[7];
		state[8] = p[8];

		#pragma unroll
		for (int i = 9; i<25; i++) state[i] = make_uint2(0, 0);

		keccak_1600(state);

		state[0] ^= p[9];
		state[1] ^= p[10];
		state[2] ^= p[11];
		state[3] ^= p[12];
		state[4] ^= p[13];
		state[5] ^= p[14];
		state[6] ^= p[15];
		state[7] ^= vectorize(0x05);
		state[8] ^= vectorize(1ULL << 63);

		keccak_1600(state);
		keccak_1600(state);

		if (devectorize(state[6]) <= pTarget[15]) *resNounce = nonce;
	}
}

__host__ void skein1024_cpu_init(int thr_id, int threads)
{
}

__host__ uint64_t skein1024_cpu_hash(int thr_id, int threads, uint64_t startNounce, int order, int threadsperblock)
{
	uint64_t result = 0xffffffffffffffff;
	hipMemset(d_SKNonce[thr_id], 0xff, sizeof(uint64_t));

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	skein1024_gpu_hash_35 << <grid, block >> >(threads, startNounce, d_SKNonce[thr_id]);
	hipMemcpy(d_sknounce[thr_id], d_SKNonce[thr_id], sizeof(uint64_t), hipMemcpyDeviceToHost);

	MyStreamSynchronize(NULL, order, thr_id);

	result = *d_sknounce[thr_id];
	return result;
}

__host__ void skein1024_setBlock(void *pdata)
{
	uint2 hv[17];
	uint64_t t[3];
	uint64_t h[17];
	uint64_t p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15;

	uint64_t cpu_skein_ks_parity = 0x5555555555555555;
	h[16] = cpu_skein_ks_parity;
	for (int i = 0; i<16; i++) {
		h[i] = cpu_SKEIN1024_IV_1024[i];
		h[16] ^= h[i];
	}
	uint64_t* alt_data = (uint64_t*)pdata;
	/////////////////////// round 1 //////////////////////////// should be on cpu => constant on gpu
	p0 = alt_data[0];
	p1 = alt_data[1];
	p2 = alt_data[2];
	p3 = alt_data[3];
	p4 = alt_data[4];
	p5 = alt_data[5];
	p6 = alt_data[6];
	p7 = alt_data[7];
	p8 = alt_data[8];
	p9 = alt_data[9];
	p10 = alt_data[10];
	p11 = alt_data[11];
	p12 = alt_data[12];
	p13 = alt_data[13];
	p14 = alt_data[14];
	p15 = alt_data[15];
	t[0] = 0x80; // ptr  
	t[1] = 0x7000000000000000; // etype
	t[2] = 0x7000000000000080;

	p0 += h[0];
	p1 += h[1];
	p2 += h[2];
	p3 += h[3];
	p4 += h[4];
	p5 += h[5];
	p6 += h[6];
	p7 += h[7];
	p8 += h[8];
	p9 += h[9];
	p10 += h[10];
	p11 += h[11];
	p12 += h[12];
	p13 += h[13] + t[0];
	p14 += h[14] + t[1];
	p15 += h[15];

	for (int i = 1; i < 21; i += 2)
	{
		Round1024_host(p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, 0);
		Round1024_host(p0, p9, p2, p13, p6, p11, p4, p15, p10, p7, p12, p3, p14, p5, p8, p1, 1);
		Round1024_host(p0, p7, p2, p5, p4, p3, p6, p1, p12, p15, p14, p13, p8, p11, p10, p9, 2);
		Round1024_host(p0, p15, p2, p11, p6, p13, p4, p9, p14, p1, p8, p5, p10, p3, p12, p7, 3);

		p0 += h[(i + 0) % 17];
		p1 += h[(i + 1) % 17];
		p2 += h[(i + 2) % 17];
		p3 += h[(i + 3) % 17];
		p4 += h[(i + 4) % 17];
		p5 += h[(i + 5) % 17];
		p6 += h[(i + 6) % 17];
		p7 += h[(i + 7) % 17];
		p8 += h[(i + 8) % 17];
		p9 += h[(i + 9) % 17];
		p10 += h[(i + 10) % 17];
		p11 += h[(i + 11) % 17];
		p12 += h[(i + 12) % 17];
		p13 += h[(i + 13) % 17] + t[(i + 0) % 3];
		p14 += h[(i + 14) % 17] + t[(i + 1) % 3];
		p15 += h[(i + 15) % 17] + (uint64_t)i;

		Round1024_host(p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, 4);
		Round1024_host(p0, p9, p2, p13, p6, p11, p4, p15, p10, p7, p12, p3, p14, p5, p8, p1, 5);
		Round1024_host(p0, p7, p2, p5, p4, p3, p6, p1, p12, p15, p14, p13, p8, p11, p10, p9, 6);
		Round1024_host(p0, p15, p2, p11, p6, p13, p4, p9, p14, p1, p8, p5, p10, p3, p12, p7, 7);

		p0 += h[(i + 1) % 17];
		p1 += h[(i + 2) % 17];
		p2 += h[(i + 3) % 17];
		p3 += h[(i + 4) % 17];
		p4 += h[(i + 5) % 17];
		p5 += h[(i + 6) % 17];
		p6 += h[(i + 7) % 17];
		p7 += h[(i + 8) % 17];
		p8 += h[(i + 9) % 17];
		p9 += h[(i + 10) % 17];
		p10 += h[(i + 11) % 17];
		p11 += h[(i + 12) % 17];
		p12 += h[(i + 13) % 17];
		p13 += h[(i + 14) % 17] + t[(i + 1) % 3];
		p14 += h[(i + 15) % 17] + t[(i + 2) % 3];
		p15 += h[(i + 16) % 17] + (uint64_t)(i + 1);

	}

	h[0] = p0^alt_data[0];
	h[1] = p1^alt_data[1];
	h[2] = p2^alt_data[2];
	h[3] = p3^alt_data[3];
	h[4] = p4^alt_data[4];
	h[5] = p5^alt_data[5];
	h[6] = p6^alt_data[6];
	h[7] = p7^alt_data[7];
	h[8] = p8^alt_data[8];
	h[9] = p9^alt_data[9];
	h[10] = p10^alt_data[10];
	h[11] = p11^alt_data[11];
	h[12] = p12^alt_data[12];
	h[13] = p13^alt_data[13];
	h[14] = p14^alt_data[14];
	h[15] = p15^alt_data[15];
	h[16] = cpu_skein_ks_parity;
	for (int i = 0; i<16; i++) { h[16] ^= h[i]; }
	for (int i = 0; i<17; i++) { hv[i] = lohi_host(h[i]); } //will slow down things


	uint2 cpu_Message[27];
	for (int i = 0; i<27; i++) { cpu_Message[i] = lohi_host(alt_data[i]); } //might slow down things

	hipMemcpyToSymbol(HIP_SYMBOL(c_hv), hv, sizeof(hv), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(uMessage), cpu_Message, sizeof(cpu_Message), 0, hipMemcpyHostToDevice);
}

__host__ void sk1024_keccak_cpu_init(int thr_id, int threads)
{
	hipMalloc(&d_SKNonce[thr_id], sizeof(uint64_t));
	hipHostMalloc(&d_sknounce[thr_id], 1 * sizeof(uint64_t));
}


__host__ void sk1024_set_Target(const void *ptarget)
{
	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), ptarget, 16 * sizeof(uint64_t), 0, hipMemcpyHostToDevice);
}

